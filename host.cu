#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdint.h> 
#include <stdlib.h>

#define CHECK(call){\
    const hipError_t error = call;\
    if (error != hipSuccess)\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error, hipGetErrorString(error));\
        exit(EXIT_FAILURE);\
}
struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start, 0);                                                                 
		hipEventSynchronize(start);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

typedef struct {
     unsigned char red,green,blue;
} PPMPixel;

typedef struct {
     int x, y;
     PPMPixel *data;
} PPMImage;

#define CREATOR "QUOCBAO"
#define RGB_COMPONENT_COLOR 255

static PPMImage *readPPM(const char *filename)
{
    char buff[16];
    PPMImage *img;
    FILE *fp;
    int c, rgb_comp_color;
    //open PPM file for reading
    fp = fopen(filename, "rb");
    if (!fp) {
        fprintf(stderr, "Unable to open file '%s'\n", filename);
        exit(1);
    }

    //read image format
    if (!fgets(buff, sizeof(buff), fp)) {
        perror(filename);
        exit(1);
    }

    //check the image format
    if (buff[0] != 'P' || buff[1] != '6') {
         fprintf(stderr, "Invalid image format (must be 'P6')\n");
         exit(1);
    }

    //allocate memory form image
    img = (PPMImage *)malloc(sizeof(PPMImage));
    if (!img) {
         fprintf(stderr, "Unable to allocate memory\n");
         exit(1);
    }

    //check for comments
    c = getc(fp);
    while (c == '#') {
    while (getc(fp) != '\n') ;
         c = getc(fp);
    }

    ungetc(c, fp);
    //read image size information
    if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
         fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
         exit(1);
    }

    //read rgb component
    if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
         fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
         exit(1);
    }

    //check rgb component depth
    if (rgb_comp_color!= RGB_COMPONENT_COLOR) {
         fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
         exit(1);
    }

    while (fgetc(fp) != '\n') ;
    //memory allocation for pixel data
    img->data = (PPMPixel*)malloc(img->x * img->y * sizeof(PPMPixel));

    if (!img) {
         fprintf(stderr, "Unable to allocate memory\n");
         exit(1);
    }

    //read pixel data from file
    if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
         fprintf(stderr, "Error loading image '%s'\n", filename);
         exit(1);
    }

    fclose(fp);
    return img;
}

void writePPM(const char *filename, PPMImage *img)
{
    FILE *fp;
    // open file for output
    fp = fopen(filename, "wb");
    if (!fp) {
         fprintf(stderr, "Unable to open file '%s'\n", filename);
         exit(1);
    }

    // write the header file
    // image format
    fprintf(fp, "P6\n");

    //comments
    fprintf(fp, "# Created by %s\n",CREATOR);

    //image size
    fprintf(fp, "%d %d\n",img->x,img->y);

    // rgb component depth
    fprintf(fp, "%d\n",RGB_COMPONENT_COLOR);

    // pixel data
    fwrite(img->data, 3 * img->x, img->y, fp);
    fclose(fp);
}

uint8_t * ChangeRGBtoGrayScale(PPMImage *img, int numChannels){
    // create grayscale array 
    uint8_t * grayscale_pixels;
    
    if (img){
        // allocate memory 
        grayscale_pixels = (uint8_t *)malloc(img->x*img->y*sizeof(uint8_t));
        for (int i = 0; i < img->x*img->y; i ++){
            grayscale_pixels[i] = 0.299f*img->data[i].red + 0.587f*img->data[i].green + 0.114f*img->data[i].blue;
        }
    }
    return grayscale_pixels;
}

// PPMImage * ChangeGrayScaletoRGB(uint8_t *pixels, int width, int height){
//     // create PPMImage array 
//     PPMImage * img;
//     // allocate the memory 
//     img = (PPMImage*)malloc(sizeof(PPMImage));
//     if (pixels){
//       // allocate the memory for data pixels
//       img->x = width;
//       img->y = height;
//       img->data = (PPMPixel*)malloc(width*height*sizeof(PPMPixel));
//       for (int i =0; i < width*height; i++){
//           int red = pixels[i];
//           int green = pixels[i];
//           int blue = pixels[i];
//           img->data[i].red = red;
//           img->data[i].green = green;
//           img->data[i].blue = blue;
//       }
//     }
//     return img;
// }

void writeGrayScale_Pnm(int * pixels, int width, int height, int numChannels, char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	if (numChannels == 1)
		fprintf(f, "P2\n");
	else if (numChannels == 3)
		fprintf(f, "P3\n");
	else
	{
		fclose(f);
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	fprintf(f, "%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height; i++)
		fprintf(f, "%hhu\n", pixels[i]);

	fclose(f);
}

void printPixels(PPMImage *img){
  if (img){
    for (int i = 0; i < img->x*img->y; i++){
      printf("red: %d, green: %d, blue: %d\n", img->data[i].red, img->data[i].green, img->data[i].blue);
    }
  }
}


float computeError(PPMPixel * a1, PPMPixel * a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
	{
		err += abs((int)a1[i].red - (int)a2[i].red);
		err += abs((int)a1[i].green - (int)a2[i].green);
		err += abs((int)a1[i].blue - (int)a2[i].blue);
	}
	err /= (n * 3);
	return err;
}

// void printError(uchar3 * deviceResult, uchar3 * hostResult, int width, int height)
// {
// 	float err = computeError(deviceResult, hostResult, width * height);
// 	printf("Error: %f\n", err);
// }

void printDeviceInfo()
{
	hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("CMEM: %lu bytes\n", devProv.totalConstMem);
    printf("L2 cache: %i bytes\n", devProv.l2CacheSize);
    printf("SMEM / one SM: %lu bytes\n", devProv.sharedMemPerMultiprocessor);
    printf("****************************\n");

}



int* ComputeImportanceMap(uint8_t * grayscalepixels, int width, int height){
    int x_sobel[3][3] = {{1, 0, -1}, {2, 0, -2}, {1, 0, -1}};
    int y_sobel[3][3] = {{1, 2, 1}, {0, 0, 0}, {-1, -2, -1}};
    
    int* importancemap = (int*)malloc((width)*(height)*sizeof(int));
    // detect edges in the x direction
  
    // detect edges in the y direction 
    for (int row = 0; row < height; row++){
        for(int col = 0; col < width; col ++){
          int Gx =0, Gy = 0;
          Gx = grayscalepixels[row*width + col]*x_sobel[0][0]+grayscalepixels[row*width + col + 1]*x_sobel[0][1]+grayscalepixels[row*width + col + 2]*x_sobel[0][2]+\
                    grayscalepixels[(row+1)*width + col]*x_sobel[1][0]+grayscalepixels[(row+1)*width + col + 1]*x_sobel[1][1]+grayscalepixels[(row+1)*width + col + 2]*x_sobel[1][2]+\
                    grayscalepixels[(row+2)*width + col]*x_sobel[2][0]+grayscalepixels[(row+2)*width + col + 1]*x_sobel[2][1]+grayscalepixels[(row+2)*width + col + 2]*x_sobel[2][2];

          Gy = grayscalepixels[row*width + col]*y_sobel[0][0]+grayscalepixels[row*width + col + 1]*y_sobel[0][1]+grayscalepixels[row*width + col + 2]*y_sobel[0][2]+\
                    grayscalepixels[(row+1)*width + col]*y_sobel[1][0]+grayscalepixels[(row+1)*width + col + 1]*y_sobel[1][1]+grayscalepixels[(row+1)*width + col + 2]*y_sobel[1][2]+\
                    grayscalepixels[(row+2)*width + col]*y_sobel[2][0]+grayscalepixels[(row+2)*width + col + 1]*y_sobel[2][1]+grayscalepixels[(row+2)*width + col + 2]*y_sobel[2][2];

          importancemap[row*width + col] = abs(Gx) + abs(Gy);
        }
    }
    return importancemap;
}

int findMin(int a, int b){
    if (a <= b)
        return a;
    else 
        return b;
}

int findMinIndex(int *arr, int start, int end){
    int min_index = start;
    int min_value = arr[start];

    for (int i = start + 1; i <= end; i++){
        if(arr[i] < min_value){
            min_value = arr[i];
            min_index = i;
        }
    }
    return min_index;
}

void FindSeam(int* importantmap, int width, int height, int seam[]){
    int **cumulative_energy = new int*[height];
    
    for(int i = 0; i < height; i++){
      cumulative_energy[i] = new int[width];
    }

    // copy the first row of the important map to the cumulative energy map
    for (int j=0; j < height; j++){
        cumulative_energy[0][j] = importantmap[j];
    }

    // Create a cumulative energy map 
    for (int i = 1; i < height; i++){
        for (int j =0; j< width; j++){

            if ( j - 1 < 0){
                cumulative_energy[i][j] = importantmap[i*width+j] +\
                findMin(cumulative_energy[i-1][j], cumulative_energy[i-1][j+1]);
            }
            else if ( j + 1 > width - 1){
                cumulative_energy[i][j] = importantmap[i*width+j] +\
                findMin(cumulative_energy[i-1][j -1], cumulative_energy[i-1][j]);
            }
            else{
                cumulative_energy[i][j] = importantmap[i*width+j] +\
            findMin(findMin(cumulative_energy[i-1][j -1], cumulative_energy[i-1][j]), cumulative_energy[i-1][j+1]);
            }
        }
    }

    // find minumum cumulative energy in the last row 
    int min_energy_index = findMinIndex(cumulative_energy[height - 1], 0, width -1);
    
    seam[height - 1] = min_energy_index;

    // back tracking the seam 
    for (int i = height - 2; i >= 0; i--){
        int min_index = findMinIndex(cumulative_energy[i], max(0, min_energy_index -1), min(width -1, min_energy_index +1));
        min_energy_index = min_index;
        seam[i] = min_energy_index;
    }
}


PPMImage* SeamCarvingHost(PPMImage *img, int width, int height, int re_width){
    int n = width - re_width;

    // copy temp image to not change original image 
    PPMImage * temp_img = (PPMImage*)malloc(sizeof(PPMImage));
    temp_img->x = width;
    temp_img->y = height;
    temp_img->data = (PPMPixel*)malloc(width*height*sizeof(PPMPixel));
    for (int i = 0; i < height; i++){
      for(int j =0; j < width; j++){
        temp_img->data[i*width + j].red = img->data[i*width + j].red;
        temp_img->data[i*width + j].green = img->data[i*width + j].green;
        temp_img->data[i*width + j].blue = img->data[i*width + j].blue;
      }
    }

    // seam carving job
    for (int i = 1; i <= n; i ++){
      // change rgb image to grayscale image 
      uint8_t* grayscale_pixels = ChangeRGBtoGrayScale(temp_img, 3);

      // char grayscale[] = "grayscale.ppm";
      // writeGrayScale_Pnm(grayscale_pixels, width, height,1, grayscale);

      // find energy map 
      int * importance_map = ComputeImportanceMap(grayscale_pixels, width, height);

      // change energy_map into image
      // uint8_t* energy_map = (uint8_t *)malloc(width*height*sizeof(uint8_t));
      // for (int i =0; i < height; i++ ){
      //   for (int j =0; j < width; j++){
      //       energy_map[i*width + j] = importance_map[i*width + j];
      //   }
      // }
      // char energy[] = "energy.ppm";
      // writeGrayScale_Pnm(energy_map, width, height,1, energy);

      // find seam 
      int seam[1000];
      FindSeam(importance_map, width, height, seam);

      // create new image after removing seam
      PPMImage * new_img = (PPMImage*)malloc(sizeof(PPMImage));
      new_img->data = (PPMPixel*)malloc(height*(width-1)*sizeof(PPMPixel));
      new_img->x = width -1;
      new_img->y = height;

      // removal seam 
      for(int y = 0; y < height; y++){
        if (seam[y] == 0){
          for(int j = 0; j < width-1; j ++){
              new_img->data[y*(width - 1) + j].red = temp_img->data[y*width + j+1].red;
              new_img->data[y*(width - 1) + j].green = temp_img->data[y*width + j+1].green;
              new_img->data[y*(width - 1) + j].blue = temp_img->data[y*width + j+1].blue;
          }
        }
        else if ( seam[y]>0 && seam[y] < width -1){
          for (int j = 0; j < seam[y]; j ++){
            new_img->data[y*(width -1) + j ].red = temp_img->data[y*width + j].red;
            new_img->data[y*(width -1) + j ].green = temp_img->data[y*width + j].green;
            new_img->data[y*(width -1) + j ].blue = temp_img->data[y*width + j].blue;
          }
          for (int j = seam[y]; j < width -1; j ++){
            new_img->data[y*(width -1) + j].red = temp_img->data[y*width + j + 1].red;
            new_img->data[y*(width -1) + j].green = temp_img->data[y*width + j + 1].green;
            new_img->data[y*(width -1) + j].blue = temp_img->data[y*width + j + 1].blue;
          }
        }
        else if ( seam[y] == width -1){
          for (int j= 0 ; j < width -1; j ++){
            new_img->data[y*(width -1) + j ].red = temp_img->data[y*width + j].red;
            new_img->data[y*(width -1) + j ].green = temp_img->data[y*width + j].green;
            new_img->data[y*(width -1) + j ].blue = temp_img->data[y*width + j].blue;
          }
        }
      }

      // update original image size 
      width -=1;
      free(temp_img);
      temp_img = new_img;

    }
    // write image 
    char out_rgb[] = "out_host_rgb.ppm";
    writePPM(out_rgb, temp_img);

  return temp_img;
}

__global__ void ConvertRgb2Gray_Kernel(PPMPixel * pixels, int width, int height, uint8_t * grayPic) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < height && c < width) {
        int i = r * width + c;
        grayPic[i] = 0.299f*pixels[i].red + 0.587f*pixels[i].green + 0.114f*pixels[i].blue;
    }
    __syncthreads();
}

__global__ void ComputeImportanceMap_Kernel(uint8_t * grayscalepixels, int *energy, int width, int height){
    int x_sobel[3][3] = {{1, 0, -1}, {2, 0, -2}, {1, 0, -1}};
    int y_sobel[3][3] = {{1, 2, 1}, {0, 0, 0}, {-1, -2, -1}};

    int col = threadIdx.x + blockIdx.x*blockDim.x;
    int row = threadIdx.y + blockIdx.y*blockDim.y;
  
    if ( col < width && row < height){    
      int Gx = grayscalepixels[row*width + col]*x_sobel[0][0]+grayscalepixels[row*width + col + 1]*x_sobel[0][1]+grayscalepixels[row*width + col + 2]*x_sobel[0][2]+\
                    grayscalepixels[(row+1)*width + col]*x_sobel[1][0]+grayscalepixels[(row+1)*width + col + 1]*x_sobel[1][1]+grayscalepixels[(row+1)*width + col + 2]*x_sobel[1][2]+\
                    grayscalepixels[(row+2)*width + col]*x_sobel[2][0]+grayscalepixels[(row+2)*width + col + 1]*x_sobel[2][1]+grayscalepixels[(row+2)*width + col + 2]*x_sobel[2][2];
      int Gy = grayscalepixels[row*width + col]*y_sobel[0][0]+grayscalepixels[row*width + col + 1]*y_sobel[0][1]+grayscalepixels[row*width + col + 2]*y_sobel[0][2]+\
                    grayscalepixels[(row+1)*width + col]*y_sobel[1][0]+grayscalepixels[(row+1)*width + col + 1]*y_sobel[1][1]+grayscalepixels[(row+1)*width + col + 2]*y_sobel[1][2]+\
                    grayscalepixels[(row+2)*width + col]*y_sobel[2][0]+grayscalepixels[(row+2)*width + col + 1]*y_sobel[2][1]+grayscalepixels[(row+2)*width + col + 2]*y_sobel[2][2];
      energy[row*width + col] = abs(Gx) + abs(Gy);
    }
    __syncthreads();
}

// __device__ int min3(int a, int b, int c) {
//     return min(a, min(b, c));
// }
__global__ void FindSeam_Kernel(int* importantmap, int width, int height, int* seam, int * cumulative_energy){

    // Create a cumulative energy map 
    // declare a device array to store cumulative values
    // int * cumulative_energy;
    // hipMalloc(&cumulative_energy, width*height*sizeof(int));
    
    int col = blockIdx.x*blockDim.x+threadIdx.x;
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    
    for (int i = 0; i < width; i++){
      cumulative_energy[i] = importantmap[i];
    }
  
    for (int j = 1; j < height; j++){
      // initialize for cumulative map for the first row.
      int left = (col > 0)?cumulative_energy[(j-1)*width + col -1]:INT_MAX;
      int middle = cumulative_energy[(j-1)*width + col];
      int right = (col < width-1) ? cumulative_energy[(j-1)*width +col +1]:INT_MAX;
      cumulative_energy[j*width + col] = importantmap[j*width + col] + min(min(left, middle), right);
      // __syncthreads();
    }
    
  
    // __syncthreads();
    // Find the minimum energy seam in the last row
    if ( row == height - 1){
      int min_value = cumulative_energy[row*width];
      int min_index = 0;
      for(int i = 1; i < width; i++){
        if ( cumulative_energy[row*width+i] < min_value){
          min_value = cumulative_energy[row*width+i];
          min_index = i;
        }
      }
      seam[row] = min_index;

      for (int i = height - 2; i >= 0; --i) {
          int left = (min_index > 0) ? cumulative_energy[i * width + (min_index - 1)] : INT_MAX;
          int middle = cumulative_energy[i * width + min_index];
          int right = (min_index < width - 1) ? cumulative_energy[i * width + (min_index + 1)] : INT_MAX;

          // Determine the minimum energy path
          if (left <= middle && left <= right) {
              min_index = min_index - 1;
          } else if (right <= left && right <= middle) {
              min_index = min_index + 1;
          }
          // Update the seam array
          seam[i] = min_index;
      }

    }
    // __syncthreads();
    // Free allocated memory
    // if (row == height - 1) {
    //     hipFree(cumulative_energy);
    // }
}

// CUDA kernel to remove a seam from the image
__global__ void removeSeam_Kernel(PPMPixel *inputImage, PPMPixel *outputImage, int *seam, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (col < width && row < height) {
        int seamCol = seam[row];
        // printf("Row: %d, Col: %d\n", row, col);
        if ( col < seamCol){
          outputImage[row*(width-1) + col] = inputImage[row*width+col];
        } 
        else{
          outputImage[row*(width-1)+col-1] = inputImage[row*width+col]; 
        }
    }
    // __syncthreads();
}
void SeamCarving_Kernel(PPMImage *in_host_img, PPMImage* out_host_img,int width, int height, int re_width){
    // create device image 
    PPMImage* d_in_img;
    PPMPixel* d_in_pixels;
    // Allocate d_in_img;
    hipMalloc(&d_in_img, sizeof(PPMImage));
    hipMalloc(&d_in_pixels, width*height*sizeof(PPMPixel));

    // copy data from host to device 
    hipMemcpy(d_in_img, in_host_img, sizeof(PPMImage), hipMemcpyHostToDevice);
    hipMemcpy(d_in_pixels, in_host_img->data, width*height*sizeof(PPMPixel), hipMemcpyHostToDevice);
  
    // do seam carving algorithm.
    int n = width - re_width;
    // n = 1;
    dim3 blockDim(32, 32);
    for (int times = 1; times <= n; times ++){
      dim3 gridDim((width + blockDim.x -1)/blockDim.x, (height+blockDim.y -1)/blockDim.y);

      // convert rgb image to grayscale 
      uint8_t * grayscale;
      hipMalloc(&grayscale, width*height*sizeof(uint8_t));
      ConvertRgb2Gray_Kernel<<<gridDim, blockDim>>>(d_in_pixels, width, height, grayscale);

      // test grayscale 
      // uint8_t * testgrayscale = (uint8_t*)malloc(width*height*sizeof(uint8_t));
      // hipMemcpy(testgrayscale, grayscale, sizeof(uint8_t)*width*height, hipMemcpyDeviceToHost);
      // printf("Grayscale image\n");
      // for(int i = 0; i<height; i ++){
      //   for(int j = 0; j < width; j++){
      //     printf("%d ", testgrayscale[i*width + j]);
      //   }
      //   printf("\n");
      // }


      // // Compute energy
      int* energy;
      hipMalloc(&energy, width*height*sizeof(int));
      ComputeImportanceMap_Kernel<<<gridDim, blockDim>>>(grayscale, energy, width, height);
      
      // // test energy 
      // int * testenergy = (int*)malloc(width*height*sizeof(int));
      // hipMemcpy(testenergy, energy, sizeof(int)*width*height, hipMemcpyDeviceToHost);
      // printf("Energy map\n");
      // for(int i = 0; i<height; i ++){
      //   for(int j = 0; j < width; j++){
      //     printf("%d ", testenergy[i*width + j]);
      //   }
      //   printf("\n");
      // }


      // int* temp = (int *)malloc(width*height*sizeof(int));
      // hipMemcpy(temp, energy, width*height*sizeof(int), hipMemcpyDeviceToHost);
      // char out_rgb[] = "out_device_energy.ppm";
      // writeGrayScale_Pnm(temp, width, height, 1, out_rgb);

      // // Find min seam 
      int* seam;
      hipMalloc(&seam, height*sizeof(int));

      
      int * cumulative_energy;
      hipMalloc(&cumulative_energy, width*height*sizeof(int));

      FindSeam_Kernel<<<gridDim, blockDim>>>(energy, width, height, seam, cumulative_energy);
      
      // test cumulative matrix
      // int * testcumulative = (int*)malloc(width*height*sizeof(int));
      // hipMemcpy(testcumulative, cumulative_energy, sizeof(int)*width*height, hipMemcpyDeviceToHost);
      // printf("Cumulative map\n");
      // for(int i = 0; i<height; i ++){
      //   for(int j = 0; j < width; j++){
      //     printf("%d ", testcumulative[i*width + j]);
      //   }
      //   printf("\n");
      // }

      // test seam 
      // int * resultseam = (int*)malloc(height*sizeof(int));
      // hipMemcpy(resultseam, seam, sizeof(int)*height, hipMemcpyDeviceToHost);
      // printf("Seam\n");
      // for (int i = 0; i < height; i++){
      //   printf("%d \n", resultseam[i]);
      // }
  
      // allocate device out pixels
      PPMPixel * d_out_pixels;
      hipMalloc(&d_out_pixels, height*(width-1)*sizeof(PPMPixel));
      removeSeam_Kernel<<<gridDim, blockDim>>>(d_in_pixels, d_out_pixels, seam,width, height);
      
      hipFree(d_in_pixels);
      d_in_pixels = d_out_pixels;
      // for(int i = 0; i< height; i++){
      //   for (int j = 0; j < width -1; j ++){
      //     printf("[%d %d %d] ",d_in_pixels[i*(width-1) + j].red, d_in_pixels[i*(width-1) + j].green, d_in_pixels[i*(width-1) + j].blue);
      //   }
      //   printf("\n");
      // }

      // // Update size of image
      width-=1;
      // d_in_img->x = width;
      // hipFree(d_in_pixels);
      // hipMalloc(&d_in_pixels, width*height*sizeof(PPMPixel));
      // hipMemcpy(d_in_pixels, d_out_pixels, width*height*sizeof(PPMPixel), hipMemcpyDeviceToDevice);
      
      // free memory
      // hipFree(d_out_pixels);
      hipFree(grayscale);
      hipFree(energy);
      hipFree(cumulative_energy);
      hipFree(seam);
      
    }
    
    // write image 
    out_host_img->data = (PPMPixel*)malloc(width*height*sizeof(PPMPixel));
    hipMemcpy(out_host_img->data, d_in_pixels,width*height*sizeof(PPMPixel), hipMemcpyDeviceToHost);
    out_host_img->x = width;
    out_host_img->y = height;
    // return d_out_img;
}


int main(int argc, char **argv){
    // process input arguments
    // Input arguments look like ./out.out (char*)img_name resize_width 

    if (argc != 3)
    {
        printf("The number of argument is invalid\n");
        return EXIT_FAILURE;
    }

    // printDeviceInfo();

    // read input image file
    PPMImage *original_image = readPPM(argv[1]);
    int width = original_image->x;
    int height = original_image->y;

    
    // temp_img = original_image;
    

    GpuTimer timer;
    timer.Start();
    PPMImage* host_img = SeamCarvingHost(original_image, original_image->x, original_image->y, atoi(argv[2]));
    timer.Stop();
    float time = timer.Elapsed();
    printf("Processing time use host: %f ms\n\n", time);
    // write image
    char out_rgb[] = "out_host_rgb.ppm";
    writePPM(out_rgb, host_img);



    // test data for find seam in kernel v1 function 
    PPMImage * temp_img = (PPMImage*)malloc(sizeof(PPMImage));
    temp_img->data = (PPMPixel*)malloc(width*height*sizeof(PPMPixel));

    temp_img->x = 5;
    temp_img->y = 5;
    temp_img->data = (PPMPixel*)malloc(temp_img->x*temp_img->y*sizeof(PPMPixel));

    for (int i = 0; i < temp_img->y; i ++){
      for(int j = 0; j < temp_img->x; j++){
        temp_img->data[i*temp_img->x + j].red = rand()%255 + 1;
        temp_img->data[i*temp_img->x + j].green = rand()%255 + 1;
        temp_img->data[i*temp_img->x + j].blue = rand()%255 + 1;
      }
    }

    // printf("Image\n");
    // for(int i = 0; i< temp_img->y; i++){
    //     for (int j = 0; j < temp_img->x; j ++){
    //       printf(" [%d %d %d] ",temp_img->data[i*(temp_img->x) + j].red,temp_img->data[i*(temp_img->x) + j].green,temp_img->data[i*(temp_img->x) + j].blue);
    //     }
    //     printf("\n");
    //   }

    // PPMImage * out_img = (PPMImage*)malloc(sizeof(PPMImage));
    // SeamCarving_Kernel_v1(temp_img, out_img, temp_img->x, temp_img->y, 300);


    // using kernel v1
    PPMImage* out_device_img = (PPMImage*)malloc(sizeof(PPMImage));
    GpuTimer timer_kernel;
    timer_kernel.Start();
    SeamCarving_Kernel(original_image, out_device_img, width, height, atoi(argv[2]));
    timer_kernel.Stop();
    float newtime_kernel = timer_kernel.Elapsed();
    printf("Processing time use device: %f ms\n\n", newtime_kernel);
    // write image
    char out_device_rgb[] = "out_device_rgb.ppm";
    writePPM(out_device_rgb, out_device_img);

    // compare error
    float err = computeError(host_img->data, out_device_img->data, host_img->x*host_img->y);
    printf("Compare error between host and device:\n");
    printf("Error: %f\n", err);
    

    // set blocksize 
    // dim3 blockSize(32, 32);


    // test_cuda_memory(original_image);
    // PPMImage * d_in_img;
    // CHECK(hipMalloc(&d_in_img, sizeof(PPMImage)));
    // CHECK(hipMalloc(&((d_in_img)->data), width*height*sizeof(PPMPixel)));


    // hipMemcpy(d_in_img, original_image, sizeof(PPMImage), hipMemcpyHostToDevice);
    // hipMemcpy((d_in_img)->data, original_image->data, width*height*sizeof(PPMPixel), hipMemcpyHostToDevice);


    // hipMemcpy(temp_img, d_in_img, sizeof(PPMImage), hipMemcpyDeviceToHost);
    // hipMemcpy(temp_img->data, (d_in_img)->data, sizeof(PPMPixel)*width*height, hipMemcpyDeviceToHost);




    
    // copy the non-pointer part of the struct to the device 

    // CHECK(hipMemcpy())

  
    // test_energy_map();

    return 0;
    
}